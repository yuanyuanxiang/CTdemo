#include "hip/hip_runtime.h"
#include "radon.h"
#include "hipblas.h"
#include "hipblas.h"

/* 
// ***************** radon �任��ʵ�� ************** //
   ********** �׶�ʦ����ѧ ������ʵ���� ********** //
// *************** Ԭ���飬2015��4�� *************** //
*/

#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }

#define CHECK_CUBLAS_ERRORS(err, str) if (err != HIPBLAS_STATUS_SUCCESS) { str = "CUBLAS ��������"; hipDeviceReset(); return str; }

/* ��ʼ������vec = [val, val, val, ...]
float*	vec:����ָ��
int		len:��������
float	val:������ֵ
ע�⣺������һά��block
*/
__global__ void cudaVectorAssigned(float* vec, int len, float val)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
		vec[i] = val;
}


// ����һ���������ݵ�����ĳһ��
/*
float *d_pDst		����
int AnglesNum		��������
float *d_pVec		����
int nNewRaysNum		��������
float alpha			���ű���
int column			��ǰ��
float Separation	���ؼ��
ע�⣺������һά��block
*/
__global__ void cudaCopyColumnData(float *d_pDst, int AnglesNum, float *d_pVec, int nNewRaysNum, float alpha, int column, float Separation)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	{
		float x = i * alpha;
		d_pDst[column + i * AnglesNum] = cudaLinearInterp(d_pVec, nNewRaysNum, x) * Separation;
	}
}


// ���Բ�ֵ
__device__ float cudaLinearInterp(float* pPrj, int nWidth, float x)
{
	int x1 = floor(x), x2 = x1 + 1;
	if ( x1 < 0 || x2 >= nWidth)
		return 0;
	return pPrj[x1] * (1 - x + x1) + pPrj[x2] * (x - x1);
}


// ��������,��ͼ�����ۼӵ�ֵ������d_pDst
/*
float *d_pDst			Ŀ��
int nNewRaysNum			Ŀ�곤��
int nDetectorCenter		Ŀ������
float *d_pSrc			Դ
int Width				Դ����
int nHalfWidth			Դ����
float fPixelDistance	���ؼ��
*/
__global__ void cudaCopyTempData(float *d_pDst, int nNewRaysNum, int nDetectorCenter, float *d_pSrc, int Width, int nHalfWidth, float fPixelDistance)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < nNewRaysNum)
	{
		int s = (i - nDetectorCenter) + nHalfWidth;
		if (s >= 0 && s < Width)
			d_pDst[i] = d_pSrc[s];
	}
}


// radon�任�ĺ˺���
/*
float* d_pSrc, int Width, int Height			Դ�Ϳ�ȡ��߶�
float* d_pDst,int NewWidth, int NewHeight		Ŀ��Ϳ�ȡ��߶�
int Xmin, int Ymin								Ŀ������½Ƕ���
float cos_theta, float sin_theta				��ת�Ƕȵ�������
*/
__global__ void cudaRotateKernel(float* d_pSrc, int Width, int Height, float* d_pDst, int NewWidth, int NewHeight, int Xmin, int Ymin, float cos_theta, float sin_theta)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; //i��
	int j = blockIdx.y * blockDim.y + threadIdx.y; //j��
	if (i < NewHeight && j < NewWidth)
	{
		float x = float(j + Xmin);
		float y = float(i + Ymin);
		cudaPositionTransform(x, y, cos_theta, -sin_theta);
		*(d_pDst + j + i * NewWidth) = cudaGetPositionValue(x, y, d_pSrc, Width, Height);
	}
}


// �������ã�����ͶӰͼ��
/*
float* h_pDst				ͶӰͼ��
int RaysNum					ÿ�Ƕ�������
int AnglesNum				�����Ƕȸ���
 ***��������������2015.4.17����***
float pixel_separation		���ؼ��
float angle_separation		�Ƕȼ��
BYTE* h_pSrc				ԭʼͼ��
int Width					ԭͼ���
int Height					ԭͼ�߶�
int Rowlen					ԭͼ���ֽ���
*/
__host__ const char* cudaRadon(float* h_pDst, int RaysNum, int AnglesNum, float pixel_separation, float angle_separation, BYTE* h_pSrc, int Width, int Height, int Rowlen)
{
	// cuda ���󼰴�����Ϣ
	hipError_t error;
	const char* errstr = 0;
	hipDeviceProp_t deviceProp;
	int devID;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, errstr);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, errstr);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, errstr);
	// ����������� < 1.1
	if (deviceProp.major < 1) return "��������С��1.0.";
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return "��������С��1.1.";

	// �豸������
	BYTE *d_pSrc = NULL;
	float *d_pDst = NULL;
	// ��������
	error = hipMalloc((void **)&d_pSrc, Height * Rowlen * sizeof(BYTE));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pSrc, h_pSrc, Height * Rowlen * sizeof(BYTE), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_pDst, RaysNum * AnglesNum * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pDst, h_pDst, RaysNum * AnglesNum * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	
	// ������cublas�����Ĳ���
	float fAlpha = 1.0f, fBeta  = 0.0f;
	hipblasHandle_t cubHandle;
	hipblasStatus_t cubStatus;
	cubStatus = cublasInit();
	CHECK_CUBLAS_ERRORS(cubStatus, errstr);
	cubStatus = hipblasCreate(&cubHandle);
	CHECK_CUBLAS_ERRORS(cubStatus, errstr);
	hipblasOperation_t cubTrans = HIPBLAS_OP_N;
	// �ȷŴ�ͼ��
	float *d_pZoom = NULL;
	// �Ŵ���
	float ZoomRate = 1.f / pixel_separation;
	int zoom_NewWidth = ZoomRate * Width;
	int zoom_NewHeight = ZoomRate * Height;
	d_pZoom = cudaImageZoom(d_pZoom, zoom_NewWidth, zoom_NewHeight, d_pSrc, Width, Height, Rowlen);
	Width = zoom_NewWidth;
	Height = zoom_NewHeight;
	Rowlen = Width;
	// ����ͼ��Ŵ�֮����������
	int nNewRaysNum = ComputeRaysNum(Width, Height);
	float *d_pTemp = NULL,	//�豸���м����ݣ�����nNewRaysNum
		*d_pOnes = NULL,	//�豸�˵�λ����������NewHeight
		*d_pWidth_add = NULL;

	error = hipMalloc((void **)&d_pTemp, nNewRaysNum * sizeof(float));
	CHECK_ERRORS(error, errstr);
	float density = 1.f * nNewRaysNum / RaysNum;
	// ��ĳ�������߻��֣�index��ʾ�ڼ����Ƕ�
	for (int index = 0; index < AnglesNum; ++index)
	{
		// ԭʼͼ���ĸ����������
		float x1, x2, x3, x4, y1, y2, y3, y4;
		x1 = 0;							y1 = 0;
		x2 = float(Width - 1);			y2 = 0;
		x3 = x2;						y3 = float(Height - 1);
		x4 = x1;						y4 = y3;
		// �ĸ�����˳ʱ����ת
		float angle = index * angle_separation;
		// 2015.5.19 Ϊ�˼��ټ��㣬�����Ǻ����ŵ�ѭ������
		float cos_theta = cos(angle);
		float sin_theta = sin(angle);
		PositionTransform(x1, y1, cos_theta, sin_theta);
		PositionTransform(x2, y2, cos_theta, sin_theta);
		PositionTransform(x3, y3, cos_theta, sin_theta);
		PositionTransform(x4, y4, cos_theta, sin_theta);
		int Xmin, Xmax, Ymin, Ymax;
		Xmin = (int)floor(FindMinBetween4Numbers(x1, x2, x3, x4));
		Xmax = (int)floor(FindMaxBetween4Numbers(x1, x2, x3, x4));
		Ymin = (int)floor(FindMinBetween4Numbers(y1, y2, y3, y4));
		Ymax = (int)floor(FindMaxBetween4Numbers(y1, y2, y3, y4));
		// ��ͼ���ȡ��߶ȵı仯
		int NewWidth = Xmax - Xmin + 1;
		int NewHeight = Ymax - Ymin + 1;
		// �����ڴ棬������NewWidth * NewHeight
		float *d_pRotatedBits = NULL;
		error = hipMalloc((void **)&d_pRotatedBits, NewHeight * NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);
		error = hipMemset(d_pRotatedBits, 0, NewHeight * NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);

		// ********** Kernel **********
		dim3 dimBlock(16, 16);
		dim3 dimGrid((NewHeight + dimBlock.x - 1) / dimBlock.x, (NewWidth + dimBlock.y - 1) / dimBlock.y);
		cudaRotateKernel<<<dimGrid, dimBlock>>>(d_pZoom, Width, Height, d_pRotatedBits, NewWidth, NewHeight, Xmin, Ymin, cos_theta, sin_theta);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);

		//ͼ�����ۼ�
		error = hipMalloc((void **)&d_pOnes, NewHeight * sizeof(float));
		CHECK_ERRORS(error, errstr);
		error = hipMalloc((void **)&d_pWidth_add, NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);
		//�˺�����<<<1, N>>>�߳����ޣ�ֻ�ܴ���512��С.�������ÿ��blockʹ��ThreadsNum���̣߳�������grid
		int ThreadsNum = 256;
		cudaVectorAssigned<<<(NewHeight + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pOnes, NewHeight, 1.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cudaVectorAssigned<<<(nNewRaysNum + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pTemp, nNewRaysNum, 0.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cudaVectorAssigned<<<(NewWidth + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pWidth_add, NewWidth, 0.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cubStatus = hipblasSgemv(cubHandle, cubTrans, NewWidth, NewHeight, &fAlpha, d_pRotatedBits, NewWidth, d_pOnes, 1, &fBeta, d_pWidth_add, 1);
		CHECK_CUBLAS_ERRORS(cubStatus, errstr);
		hipFree(d_pOnes);
		hipFree(d_pRotatedBits);
		// ̽�������ĺ�ͼ���ȵ�����
		int nDetectorCenter = (nNewRaysNum + 1) / 2;
		int nHalfWidth = (NewWidth + 1) / 2;
		cudaCopyTempData<<<(nNewRaysNum + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pTemp, nNewRaysNum, nDetectorCenter, d_pWidth_add, NewWidth, nHalfWidth, pixel_separation);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cudaCopyColumnData<<<(RaysNum + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pDst, AnglesNum, d_pTemp, nNewRaysNum, density, index, pixel_separation);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		hipFree(d_pWidth_add);
	}
	error = hipMemcpy(h_pDst, d_pDst, RaysNum * AnglesNum * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, errstr);
	hipFree(d_pTemp);
	hipFree(d_pSrc);
	hipFree(d_pDst);
	hipblasDestroy(cubHandle);
	cublasShutdown();
	hipDeviceReset();
	return errstr;
}


// ͼ��Ŵ�ĺ˺���
/*
float* d_pDst, float wRatio, float hRatio, int NewWidth, int NewHeight		Ŀ�꼰�Ŵ��ʡ�������Ϣ
BYTE* d_pSrc, int Width, int Height, int Rowlen								ԭͼ���ȡ��߶ȼ�ÿ�и�����
*/
__global__ void cudaZoomKernel(float* d_pDst, float wRatio, float hRatio, int NewWidth, int NewHeight, BYTE* d_pSrc, int Width, int Height, int Rowlen)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x; //j��
	int i = blockIdx.y * blockDim.y + threadIdx.y; //i��
	if ( i < NewWidth && j < NewHeight)
		d_pDst[i + j * NewWidth] = cudaGetImagePixel(i * wRatio, j * wRatio, d_pSrc, Width, Height, Rowlen);
}


// ��ͼ����зŴ�
/*
float* pDst		Ŀ������
int NewWidth	Ŀ����
int NewHeight	Ŀ��߶�
BYTE* pSrc		ԭͼ����
int Width		ԭͼ���
int Height		ԭͼ�߶�
int Rowlen		ԭͼÿ���ֽ���
 ** ע�� ** ���뽫d_pDst��Ϊ������ء�
*/
__host__ float* cudaImageZoom(float* d_pDst, int NewWidth, int NewHeight, BYTE* d_pSrc, int Width, int Height, int Rowlen)
{
	// ���Ŀ��ǿգ����ؿա�
	if (d_pDst != NULL)
		return NULL;
	// ��������ڴ�ʧ�ܣ����ؿ�
	if(hipMalloc((void **)&d_pDst, NewWidth * NewHeight * sizeof(float)))
		return NULL;
	// ͼ���ȡ��߶����ű���
	float wRatio = 1.f * Width / NewWidth;
	float hRatio = 1.f * Height / NewHeight;
	// ���ú˺���
	dim3 dimBlock(16, 16);
	dim3 dimGrid((NewHeight + dimBlock.x - 1) / dimBlock.x, (NewWidth + dimBlock.y - 1) / dimBlock.y);
	cudaZoomKernel<<<dimGrid, dimBlock>>>(d_pDst, wRatio, hRatio, NewWidth, NewHeight, d_pSrc, Width, Height, Rowlen);
	hipError_t error = hipGetLastError();

	return d_pDst;
}


// ��ȡͼ�����x�У�y�е�����ֵ��
__device__ float cudaGetImagePixel(int x, int y, BYTE* pSrc, int Width, int Height, int Rowlen)
{
	if (x < 0 || x >= Width || y < 0 || y >= Height)
		return 0;
	return (float)*(pSrc + x + y * Rowlen);
}


// ��ȡ����(x, y)��������ֵ������˫���Բ�ֵ��
__device__ float cudaGetImagePixel(float x, float y, BYTE* pSrc, int Width, int Height, int Rowlen)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = cudaGetImagePixel(x1, y1, pSrc, Width, Height, Rowlen) * (1 - x + x1) 
		+ cudaGetImagePixel(x2, y2, pSrc, Width, Height, Rowlen) * (x - x1);
	Ans2 = cudaGetImagePixel(x4, y4, pSrc, Width, Height, Rowlen) * (1 - x + x4) 
		+ cudaGetImagePixel(x3, y3, pSrc, Width, Height, Rowlen) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


// ��ȡ����x�У�y�е�����ֵ��
__device__ float cudaGetPositionValue(int x, int y, float* pSrc, int Width, int Height)
{
	if (x < 0 || x >= Width || y < 0 || y >= Height)
		return 0;
	return (float)*(pSrc + x + y * Width);
}


// ��ȡ����(x, y)��������ֵ������˫���Բ�ֵ��
__device__ float cudaGetPositionValue(float x, float y, float* pSrc, int Width, int Height)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = cudaGetPositionValue(x1, y1, pSrc, Width, Height) * (1 - x + x1) + cudaGetPositionValue(x2, y2, pSrc, Width, Height) * (x - x1);
	Ans2 = cudaGetPositionValue(x4, y4, pSrc, Width, Height) * (1 - x + x4) + cudaGetPositionValue(x3, y3, pSrc, Width, Height) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


// �豸�ˣ�������(x, y)��תangle�Ƕȡ�
__device__ void cudaPositionTransform(float &x, float &y, float theta)
{
	float cos_theta = cos(theta);
	float sin_theta = sin(theta);
	float x_temp = x * cos_theta - y * sin_theta;
	y = x * sin_theta + y * cos_theta;
	x = x_temp;
}


// �豸�ˣ������޸�Ϊ�Ƕȵ����Һ����ҡ�
__device__ void cudaPositionTransform(float &x, float &y, float cos_theta, float sin_theta)
{
	float x_temp = x * cos_theta - y * sin_theta;
	y = x * sin_theta + y * cos_theta;
	x = x_temp;
}