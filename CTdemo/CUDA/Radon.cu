#include "hip/hip_runtime.h"
#include "radon.h"
#include "hipblas.h"
#include "hipblas.h"

/* 
// ***************** radon �任��ʵ�� ************** //
   ********** �׶�ʦ����ѧ ������ʵ���� ********** //
// *************** Ԭ���飬2015��4�� *************** //
*/

#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }

#define CHECK_CUBLAS_ERRORS(err, str) if (err != HIPBLAS_STATUS_SUCCESS) { str = "CUBLAS ��������"; hipDeviceReset(); return str; }

/* ��ʼ������vec = [val, val, val, ...]
float*	vec:����ָ��
int		len:��������
float	val:������ֵ
ע�⣺������һά��block
*/
__global__ void cudaVectorAssigned(float* vec, int len, float val)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < len)
		vec[i] = val;
}


// ����һ���������ݵ�����ĳһ��
/*
float *d_pDst		����
int RaysNum			������
int AnglesNum		������
float *d_pVec		����
int ImageDiag		ͼ��Խ���/��������
int column			����������
int HalfRays		������һ��
int HalfDiag		��������һ��
float Separation	���߼��
ע�⣺������һά��block
*/
__global__ void cudaCopyColumnData(float *d_pDst, int RaysNum, int AnglesNum, float *d_pVec, int ImageDiag, int column, int HalfRays, int HalfDiag, float Separation)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int num = HalfDiag + (i - HalfRays) * Separation;
	if(i < RaysNum && 0 <= num && num < ImageDiag)
	{
		d_pDst[column + i * AnglesNum] = d_pVec[num];
	}
}


// �������ݣ�d_pDst[int ((i - nHalfDiag) / fPixelDistance + nHalfWidth)] = d_pSrc[temp];
__global__ void cudaCopyTempData(float *d_pDst, float *d_pSrc, int Width, int nImageDiag, int nHalfDiag, int nHalfWidth, float fPixelDistance)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < nImageDiag )
	{
		int temp = int ((i - nHalfDiag) / fPixelDistance + nHalfWidth);
		if (temp >= 0 && temp < Width)
			d_pDst[i] = d_pSrc[temp] * fPixelDistance;
	}
}


// radon�任�ĺ˺���
__global__ void cudaRotateKernel(float* d_pDst, float* d_pSrc, int Width, int Height, int Rowlen, int NewWidth, int NewHeight, int Xmin, int Ymin, float cos_theta, float sin_theta)
{
	/* ����������forѭ����д��CUDA�ں����ԡ�
	for (int i = 0; i < Width; ++i)
	{
		for (int j = 0; j < Height; ++j)
		{
			float x = float(i + Xmin);
			float y = float(j + Ymin);
			PositonTransform(x, y, -angle);
			*(pDst + i + j * NewRowlen) = (BYTE)GetPositionPixel(x, y, pSrc, Width, Height, Rowlen);
		}
	}
	*/
	int i = blockIdx.x * blockDim.x + threadIdx.x; //i��
	int j = blockIdx.y * blockDim.y + threadIdx.y; //j��
	if (i < NewHeight && j < NewWidth)
	{
		float x = float(j + Xmin);
		float y = float(i + Ymin);
		cudaPositonTransform(x, y, cos_theta, -sin_theta);
		*(d_pDst + j + i * NewWidth) = cudaGetPositionValue(x, y, d_pSrc, Width, Height);
	}
}


// �������ã�����ͶӰͼ��
/*
float* h_pDst				ͶӰͼ��
int RaysNum					ÿ�Ƕ�������
int AnglesNum				�����Ƕȸ���
 ***��������������2015.4.17����***
float rays_separation		���߼��
float angle_separation		�Ƕȼ��
BYTE* h_pSrc				ԭʼͼ��
int Width					ԭͼ���
int Height					ԭͼ�߶�
int Rowlen					ԭͼ���ֽ���
*/
__host__ const char* cudaRadon(float* h_pDst, int RaysNum, int AnglesNum, float rays_separation, float angle_separation, BYTE* h_pSrc, int Width, int Height, int Rowlen, float fSubPixel)
{
	// cuda ����
	hipError_t error;
	const char* errstr = 0;
	hipDeviceProp_t deviceProp;
	int devID;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, errstr);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, errstr);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, errstr);
	// ����������� < 1.1
	if (deviceProp.major < 1) return errstr;
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return errstr;

	// �豸������
	BYTE *d_pSrc = NULL;
	float *d_pDst = NULL;
	// ��������
	error = hipMalloc((void **)&d_pSrc, Height * Rowlen * sizeof(BYTE));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pSrc, h_pSrc, Height * Rowlen * sizeof(BYTE), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_pDst, RaysNum * AnglesNum * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pDst, h_pDst, RaysNum * AnglesNum * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	// ԭʼͼ��Խ��߳���
	int ImageDiag = (int)sqrt(1.0f * Width * Width + Height * Height);
	float *d_pTemp = NULL,	//�豸���м����ݣ�����ImageDiag
		*d_pOnes = NULL,	//�豸�˵�λ����������NewHeight
		*d_pTemp2 = NULL;

	error = hipMalloc((void **)&d_pTemp, ImageDiag * sizeof(float));
	CHECK_ERRORS(error, errstr);
	
	// ������cublas�����Ĳ���
	float fAlpha = 1.0f, fBeta  = 0.0f;
	hipblasHandle_t cubHandle;
	hipblasStatus_t cubStatus;
	cubStatus = cublasInit();
	CHECK_CUBLAS_ERRORS(cubStatus, errstr);
	cubStatus = hipblasCreate(&cubHandle);
	CHECK_CUBLAS_ERRORS(cubStatus, errstr);
	hipblasOperation_t cubTrans = HIPBLAS_OP_N;
	// �ȷŴ�ͼ��
	float *d_pZoom = NULL;
	// �Ŵ���
	float ZoomRate = 1.f / fSubPixel;
	d_pZoom = cudaImageZoom(d_pZoom, d_pSrc, ZoomRate * Width, ZoomRate * Height, Width, Height, Rowlen);
	Width *= ZoomRate;
	Height *= ZoomRate;
	Rowlen = Width;
	// ��ĳ�������߻��֣�index��ʾ�ڼ����Ƕ�
	for (int index = 0; index < AnglesNum; ++index)
	{
		// ԭʼͼ���ĸ����������
		float x1, x2, x3, x4, y1, y2, y3, y4;
		x1 = 0;							y1 = 0;
		x2 = float(Width - 1);			y2 = 0;
		x3 = x2;						y3 = float(Height - 1);
		x4 = x1;						y4 = y3;
		// �ĸ�����˳ʱ����ת
		float angle = index * angle_separation;
		// 2015.5.19 Ϊ�˼��ټ��㣬�����Ǻ����ŵ�ѭ������
		float cos_theta = cos(angle);
		float sin_theta = sin(angle);
		PositionTransform(x1, y1, cos_theta, sin_theta);
		PositionTransform(x2, y2, cos_theta, sin_theta);
		PositionTransform(x3, y3, cos_theta, sin_theta);
		PositionTransform(x4, y4, cos_theta, sin_theta);
		int Xmin, Xmax, Ymin, Ymax;
		Xmin = (int)floor(FindMinBetween4Numbers(x1, x2, x3, x4));
		Xmax = (int)floor(FindMaxBetween4Numbers(x1, x2, x3, x4));
		Ymin = (int)floor(FindMinBetween4Numbers(y1, y2, y3, y4));
		Ymax = (int)floor(FindMaxBetween4Numbers(y1, y2, y3, y4));
		// ��ͼ���ȡ��߶ȵı仯
		int NewWidth = Xmax - Xmin + 1;
		int NewHeight = Ymax - Ymin + 1;
		// �����ڴ棬������NewWidth * NewHeight
		float *d_pRotatedBits = NULL;
		error = hipMalloc((void **)&d_pRotatedBits, NewHeight * NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);
		error = hipMemset(d_pRotatedBits, 0, NewHeight * NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);

		// ********** Kernel **********
		dim3 dimBlock(16, 16);
		dim3 dimGrid((NewHeight + dimBlock.x - 1) / dimBlock.x, (NewWidth + dimBlock.y - 1) / dimBlock.y);
		cudaRotateKernel<<<dimGrid, dimBlock>>>(d_pRotatedBits, d_pZoom, Width, Height, Rowlen, NewWidth, NewHeight, Xmin, Ymin, cos_theta, sin_theta);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);

		//ͼ�����ۼ�
		error = hipMalloc((void **)&d_pOnes, NewHeight * sizeof(float));
		CHECK_ERRORS(error, errstr);
		error = hipMalloc((void **)&d_pTemp2, NewWidth * sizeof(float));
		CHECK_ERRORS(error, errstr);
		//�˺�����<<<1, N>>>�߳����ޣ�ֻ�ܴ���512��С.�������ÿ��blockʹ��ThreadsNum���̣߳�������grid
		int ThreadsNum = 256;
		cudaVectorAssigned<<<(NewHeight + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pOnes, NewHeight, 1.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cudaVectorAssigned<<<(ImageDiag + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pTemp, ImageDiag, 0.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cudaVectorAssigned<<<(NewWidth + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pTemp2, NewWidth, 0.f);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		cubStatus = hipblasSgemv(cubHandle, cubTrans, NewWidth, NewHeight, &fAlpha, d_pRotatedBits, NewWidth, d_pOnes, 1, &fBeta, d_pTemp2, 1);
		CHECK_CUBLAS_ERRORS(cubStatus, errstr);
		int HalfRays = (RaysNum + 1) / 2, HalfDiag = (ImageDiag + 1) / 2, HalfWidth = (NewWidth + 1) / 2;
		cudaCopyTempData<<<(NewWidth + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pTemp, d_pTemp2, NewWidth, ImageDiag, HalfDiag, HalfWidth, fSubPixel);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
		hipFree(d_pOnes);
		hipFree(d_pTemp2);
		hipFree(d_pRotatedBits);

		cudaCopyColumnData<<<(RaysNum + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(d_pDst, RaysNum, AnglesNum, d_pTemp, ImageDiag, index, HalfRays, HalfDiag, rays_separation);
		error = hipGetLastError();
		CHECK_ERRORS(error, errstr);
	}
	error = hipMemcpy(h_pDst, d_pDst, RaysNum * AnglesNum * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, errstr);
	hipFree(d_pTemp);
	hipFree(d_pSrc);
	hipFree(d_pDst);
	hipblasDestroy(cubHandle);
	cublasShutdown();
	hipDeviceReset();
	return errstr;
}


// ͼ��Ŵ�ĺ˺���
__global__ void cudaZoomKernel(float* d_pDst, BYTE* d_pSrc, float wRatio, float hRatio, int NewWidth, int NewHeight, int Width, int Height, int Rowlen)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x; //j��
	int i = blockIdx.y * blockDim.y + threadIdx.y; //i��
	if ( i < NewWidth && j < NewHeight)
		d_pDst[i + j * NewWidth] = cudaGetPositionPixel(i * wRatio, j * wRatio, d_pSrc, Width, Height, Rowlen);
}


// ��ͼ����зŴ�
/*
float* pDst		Ŀ������
BYTE* pSrc		ԭͼ����
int NewWidth	Ŀ����
int NewHeight	Ŀ��߶�
int Width		ԭͼ���
int Height		ԭͼ�߶�
int Rowlen		ԭͼÿ���ֽ���
 ** ע�� ** ���뽫d_pDst��Ϊ������ء�
*/
__host__ float* cudaImageZoom(float* d_pDst, BYTE* d_pSrc, int NewWidth, int NewHeight, int Width, int Height, int Rowlen)
{
	// ���Ŀ��ǿգ����ؿա�
	if (d_pDst != NULL)
		return NULL;
	// ��������ڴ�ʧ�ܣ����ؿ�
	if(hipMalloc((void **)&d_pDst, NewWidth * NewHeight * sizeof(float)))
		return NULL;
	// ͼ���ȡ��߶����ű���
	float wRatio = 1.f * Width / NewWidth;
	float hRatio = 1.f * Height / NewHeight;
	// ���ú˺���
	dim3 dimBlock(16, 16);
	dim3 dimGrid((NewHeight + dimBlock.x - 1) / dimBlock.x, (NewWidth + dimBlock.y - 1) / dimBlock.y);
	cudaZoomKernel<<<dimGrid, dimBlock>>>(d_pDst, d_pSrc, wRatio, hRatio, NewWidth, NewHeight, Width, Height, Rowlen);
	hipError_t error = hipGetLastError();

	return d_pDst;
}


// ��ȡͼ�����x�У�y�е�����ֵ��
__device__ float cudaGetImagePixel(int x, int y, BYTE* pSrc, int Width, int Height, int Rowlen)
{
	if (x < 0 || x >= Width || y < 0 || y >= Height)
		return 0;
	return (float)*(pSrc + x + y * Rowlen);
}


// ��ȡ����(x, y)��������ֵ������˫���Բ�ֵ��
__device__ float cudaGetPositionPixel(float x, float y, BYTE* pSrc, int Width, int Height, int Rowlen)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = cudaGetImagePixel(x1, y1, pSrc, Width, Height, Rowlen) * (1 - x + x1) 
		+ cudaGetImagePixel(x2, y2, pSrc, Width, Height, Rowlen) * (x - x1);
	Ans2 = cudaGetImagePixel(x4, y4, pSrc, Width, Height, Rowlen) * (1 - x + x4) 
		+ cudaGetImagePixel(x3, y3, pSrc, Width, Height, Rowlen) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


// ��ȡ����x�У�y�е�����ֵ��
__device__ float cudaGetIndexValue(int x, int y, float* pSrc, int Width, int Height)
{
	if (x < 0 || x >= Width || y < 0 || y >= Height)
		return 0;
	return (float)*(pSrc + x + y * Width);
}


// ��ȡ����(x, y)��������ֵ������˫���Բ�ֵ��
__device__ float cudaGetPositionValue(float x, float y, float* pSrc, int Width, int Height)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = cudaGetIndexValue(x1, y1, pSrc, Width, Height) * (1 - x + x1) + cudaGetIndexValue(x2, y2, pSrc, Width, Height) * (x - x1);
	Ans2 = cudaGetIndexValue(x4, y4, pSrc, Width, Height) * (1 - x + x4) + cudaGetIndexValue(x3, y3, pSrc, Width, Height) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


// �豸�ˣ�������(x, y)��תangle�Ƕȡ�
__device__ void cudaPositonTransform(float &x, float &y, float theta)
{
	float cos_theta = cos(theta);
	float sin_theta = sin(theta);
	float x_temp = x * cos_theta - y * sin_theta;
	y = x * sin_theta + y * cos_theta;
	x = x_temp;
}


// �豸�ˣ������޸�Ϊ�Ƕȵ����Һ����ҡ�
__device__ void cudaPositonTransform(float &x, float &y, float cos_theta, float sin_theta)
{
	float x_temp = x * cos_theta - y * sin_theta;
	y = x * sin_theta + y * cos_theta;
	x = x_temp;
}