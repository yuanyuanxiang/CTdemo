#include "hip/hip_runtime.h"

#include "PanRadon.h"
#include "hip/hip_vector_types.h"
#include "vector_functions.h"

#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }

#define CHECK_CUBLAS_ERRORS(err, str) if (err != HIPBLAS_STATUS_SUCCESS) { str = "CUBLAS ��������"; hipDeviceReset(); return str; }


// �˺�����������Ǳ��������ã����ܻ��������
__global__ void Kernel_cudaIntsections(float R, float D, float pan_u0, float pan_delta_u, float center_y, 
									   float2* pDst, float2* temp, float* pPrj, int pan_rays, int pan_angles, int angles_index, 
									   float* pSrc, int Xmin, int Ymin, int Xmax, int Ymax, int Width, int Height, int intsection_length)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (j < pan_rays)
	{
		float u = -pan_u0 + j * pan_delta_u;
		float k = -u / D;
		float c = -R * k + center_y;
		cudaIntsections(pDst, temp, pPrj, pan_rays, pan_angles, j, angles_index, 
			pSrc, Xmin, Ymin, Xmax, Ymax, Width, Height, intsection_length, k, c);
	}
}


__global__ void Kernel_cudaValuesCopy(float* pDst, float* pSrc, int Width, int Height, float cos_theta, float sin_theta, 
						float m_Ox, float m_Oy, int Xmin, int Ymin, int NewWidth, int NewHeight)
{
	int j = blockIdx.x * blockDim.x + threadIdx.x; //j��
	int i = blockIdx.y * blockDim.y + threadIdx.y; //i��
	if (i < NewWidth && j < NewHeight)
	{
		float x = float(i + Xmin);
		float y = float(j + Ymin);
		_cudaPositionTransform(x, y, cos_theta, -sin_theta, m_Ox, m_Oy);
		*(pDst + i + j * NewWidth) = cudaBiLinearInterp(pSrc, Height, Width, x, y);
	}
}


/*
float* pSrc				//ͼ������
int src_width			//ͼ����
int src_height			//ͼ��߶�
float* pDst				//ͶӰ����
int pan_angles			//ͶӰ���
int pan_rays			//ͶӰ�߶�
float scan_range		//������Χ
float R					//SO
float D					//SO'
*/
__host__ const char* cudaPanRadon(float* h_pSrc, int src_width, int src_height, float* h_pDst, int pan_angles, int pan_rays, float scan_range, float R, float D)
{
	// ���󼰴�����Ϣ
	hipError_t error;
	const char* str = 0;

	hipDeviceProp_t deviceProp;
	int devID;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, str);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, str);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, str);
	// ����������� < 1.1
	if (deviceProp.major < 1) return "��������С��1.0.";
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return "��������С��1.1.";

	// �����ڴ�
	float *d_pSrc = NULL, *d_pDst = NULL;
	error = hipMalloc((void**)&d_pSrc, src_width * src_height * sizeof(float));
	CHECK_ERRORS(error, str);
	error = hipMemcpy(d_pSrc, h_pSrc, src_width * src_height * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, str);
	error = hipMalloc((void**)&d_pDst, pan_angles * pan_rays * sizeof(float));
	CHECK_ERRORS(error, str);
	error = hipMemset(d_pDst, 0, pan_angles * pan_rays * sizeof(float));
	CHECK_ERRORS(error, str);

	float diagonal = sqrt(1.f * src_width * src_width + src_height * src_height);
	float theta_0 = acos(diagonal / 2.f / R);
	float pan_u0 = D / tan(theta_0);
	float pan_delta_u = (2.f * pan_u0) / pan_rays;
	float angles_separation = scan_range / pan_angles;
	float center_y = (src_height + 1) / 2.f;
	for (int i = 0; i < pan_angles; ++i)
	{
		int Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight;
		float theta = i * angles_separation;
		float cos_theta = cos(theta);
		float sin_theta = sin(theta);
		float *d_pCur = NULL;
		d_pCur = cudaComputeRotatedValues(d_pSrc, src_width, src_height, cos_theta, sin_theta, Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight);
		int intsection_length = NewWidth + NewHeight;
		float2 *IntSections = NULL, *temp = NULL;
		error = hipMalloc((void**)&IntSections, pan_rays * intsection_length * sizeof(float2));
		error = hipMemset(IntSections, 0, pan_rays * intsection_length * sizeof(float2));
		error = hipMalloc((void**)&temp, pan_rays * intsection_length * sizeof(float2));
		error = hipMemset(temp, 0, pan_rays * intsection_length * sizeof(float2));
		
		int ThreadsNum = 256;
		Kernel_cudaIntsections<<<(pan_rays + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(R, D, pan_u0, pan_delta_u, center_y, 
			IntSections, temp, d_pDst, pan_rays, pan_angles, i, d_pCur, Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight, intsection_length);
		error = hipGetLastError();
		CHECK_ERRORS(error, str);

		error = hipFree(temp);
		error = hipFree(IntSections);
		error = hipFree(d_pCur);
	}
	error = hipMemcpy(h_pSrc, d_pSrc, src_width * src_height * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, str);
	error = hipMemcpy(h_pDst, d_pDst, pan_angles * pan_rays * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, str);

	error = hipFree(d_pSrc);
	error = hipFree(d_pDst);
	return str;
}


float* cudaComputeRotatedValues(float *pSrc, int Width, int Height, float cos_theta, float sin_theta, int &Xmin, int &Ymin, int &Xmax, int &Ymax, int &NewWidth, int &NewHeight) 
{
	// ԭʼͼ���ĸ����������
	float x1, x2, x3, x4, y1, y2, y3, y4;
	x1 = 0;							y1 = 0;
	x2 = float(Width - 1);			y2 = 0;
	x3 = x2;						y3 = float(Height - 1);
	x4 = x1;						y4 = y3;
	// �ĸ�����˳ʱ����ת,������ԭ��
	float center_x = (Width + 1) / 2.f;
	float center_y = (Height + 1) / 2.f;
	PositionTransform(x1, y1, cos_theta, sin_theta, center_x, center_y);
	PositionTransform(x2, y2, cos_theta, sin_theta, center_x, center_y);
	PositionTransform(x3, y3, cos_theta, sin_theta, center_x, center_y);
	PositionTransform(x4, y4, cos_theta, sin_theta, center_x, center_y);
	Xmax = int(FindMaxBetween4Numbers(x1, x2, x3, x4));
	Ymax = int(FindMaxBetween4Numbers(y1, y2, y3, y4));
	Xmin = int(FindMinBetween4Numbers(y1, y2, y3, y4));
	Ymin = int(FindMinBetween4Numbers(y1, y2, y3, y4));
	// ��ͼ���ȡ��߶ȡ�ÿ���ֽ����ı仯
	NewWidth = Xmax - Xmin + 1;
	NewHeight = Ymax - Ymin + 1;
	float *pDst = NULL;
	hipMalloc((void**)&pDst, NewWidth * NewHeight * sizeof(float));
	hipMemset(pDst, 0, NewWidth * NewHeight * sizeof(float));
	
	dim3 dimBlock(16, 16);
	dim3 dimGrid((NewHeight + dimBlock.x - 1) / dimBlock.x, (NewWidth + dimBlock.y - 1) / dimBlock.y);
	Kernel_cudaValuesCopy<<<dimGrid, dimBlock>>>(pDst, pSrc, Width, Height, cos_theta, sin_theta, center_x, center_y, Xmin, Ymin, NewWidth, NewHeight);
	
	return pDst;
}


__device__ float cudaGetPositionValue(float *pSrc, int row, int col, int x, int y)
{
	if (x < 0 || x >= col || y < 0 || y >= row)
		return 0;
	return *(pSrc + x + y * col);
}


__device__ float cudaBiLinearInterp(float *pSrc, int row, int col, float x, float y)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = cudaGetPositionValue(pSrc, row, col, x1, y1) * (1 - x + x1) + cudaGetPositionValue(pSrc, row, col, x2, y2) * (x - x1);
	Ans2 = cudaGetPositionValue(pSrc, row, col, x4, y4) * (1 - x + x4) + cudaGetPositionValue(pSrc, row, col, x3, y3) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


__device__ __host__ float cudaLineGetXValue(float &k, float &c, float y)
{
	return (y - c) / k;
}


__device__ __host__ float cudaLineGetYValue(float &k, float &c, float x)
{
	return (k * x + c);
}


/*
float2* pDst			��������
float2* temp			�м�����
int Xmin				���½�x����
int Ymin				���½�y����
int Xmax				���Ͻ�x����
int Ymax				���Ͻ�y����
int intsection_length	�ཻ������
int line_index			���߱��
float &k				ֱ��б��
float &c				ֱ����y��ؾ�
*/
__device__ __host__ void cudaIntsections(float2* pDst, float2* temp, float* pPrj, int pan_rays, int pan_angles, int line_index, int angles_index, 
								float* pSrc, int Xmin, int Ymin, int Xmax, int Ymax, int Width, int Height, int intsection_length, float &k, float &c) 
{
	float2* dst_pos = pDst + line_index * intsection_length;
	float2* temp_pos = temp + line_index * intsection_length;
	int n = 0, n1, n2, s = 0;
	for (int i = Xmin; i <= Xmax; ++i)
	{
		float x = i;
		float y = cudaLineGetYValue(k, c, x);
		// ��Ϊֱ���ǵ������������Ե�y����ͼ���˾�break
		if (y < Ymin || y > Ymax)
			break;
		temp_pos[n++] = make_float2(x, y);
	}
	n1 = n;
	if (k >= 0)
	{
		for (int j = int(c); j <= Ymax; ++j)
		{
			float y = j;
			float x = cudaLineGetXValue(k, c, y);
			// ���ﲻ��ֱ��break��������ͼ���Կ���
			if (x < Xmin || x > Xmax)
				continue;
			temp_pos[n++] = make_float2(x, y);
		}
	}
	else
	{
		for (int j = int(c); j >= Ymin; --j)
		{
			float y = j;
			float x = cudaLineGetXValue(k, c, y);
			// ���ﲻ��ֱ��break��������ͼ���Կ���
			if (x < Xmin || x > Xmax)
				continue;
			temp_pos[n++] = make_float2(x, y);
		}
	}
	n2 = n - n1;
	// ���������������������
	if (n1 >= n2)
	{
		for (int i = 0; i < n2; ++i)
		{
			if (temp_pos[i].x < temp_pos[n1 + i].x)
			{
				dst_pos[s++] = temp_pos[i];
			}
			else
			{
				dst_pos[s++] = temp_pos[n1 + i];
			}
		}
		for (int i = n2; i < n1; ++i)
		{
			dst_pos[s++] = temp_pos[i];
		}
	}
	else
	{
		for (int i = 0; i < n1; ++i)
		{
			if (temp_pos[i].x < temp_pos[n1 + i].x)
			{
				dst_pos[s++] = temp_pos[i];
			}
			else
			{
				dst_pos[s++] = temp_pos[n1 + i];
			}
		}
		for (int i = n1; i < n2; ++i)
		{
			dst_pos[s++] = temp_pos[n1 + i];
		}
	}
	
	float sum = 0.f;
	for (int i = 0; i < s - 1; ++i)
	{
		int x = dst_pos[i].x - Xmin;
		int y = dst_pos[i].y - Ymin;
		if (0 <= x && x < Width && 0 <= y && y < Height)
			sum += cudaDistance(dst_pos[i], dst_pos[i + 1]) * pSrc[x + y * Width];
	}
	pPrj[angles_index + line_index * pan_angles] = sum;
}


/*
float2* Intsections			��������
float *pDst					ͶӰ����
float *pSrc					ԭʼͼ��
int pan_angles				�����Ƕ���
int intsection_length		�ཻ������
int angles_index			��ǰ�Ƕ�
int line_index				���߱��
int Xmin, int Ymin			ͼ�����½�����
int Width, int Height		ͼ������߶�
*/
__device__ void cudaIntergrate(float2* Intsections, float *pDst, float *pSrc, int pan_angles, int intsection_length, int angles_index, int line_index, int Xmin, int Ymin, int Width, int Height)
{
	float2* Pos = Intsections + line_index * intsection_length;
	float s = 0.f;
	for (int i = 0; i < intsection_length - 1; ++i)
	{
		int x = Pos[i].x - Xmin;
		int y = Pos[i].y - Ymin;
		if (0 <= x && x < Width && 0 <= y && y < Height)
			s += cudaDistance(Pos[i], Pos[i + 1]) * pSrc[x + y * Width];
	}
	pDst[angles_index + line_index * pan_angles] = s;
}


__device__ __host__ float cudaDistance(const float2 &a, const float2 &b)
{
	return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
}


/* ������ԭ����ת�任.*/
__device__ void _cudaPositionTransform(float &x, float &y, float cos_theta, float sin_theta)
{
	float x_temp = x * cos_theta - y * sin_theta;
	y = x * sin_theta + y * cos_theta;
	x = x_temp;
}


/* �����ƹ̶�����ת�任.*/
__device__ void _cudaPositionTransform(float &x, float &y, float cos_theta, float sin_theta, float x0, float y0)
{
	float delta_x = x - x0;
	float delta_y = y - y0;
	_cudaPositionTransform(delta_x, delta_y, cos_theta, sin_theta);
	x = x0 + delta_x;
	y = y0 + delta_y;
}