#include "hip/hip_runtime.h"

#include "PanRadon.h"
#include <algorithm>

/*
float* pSrc				//ͼ������
int src_width			//ͼ����
int src_height			//ͼ��߶�
float* pDst				//ͶӰ����
int pan_angles			//ͶӰ���
int pan_rays			//ͶӰ�߶�
float scan_range		//������Χ
float R					//SO
float D					//SO'
*/
void PanRadon(float* pSrc, int src_width, int src_height, float* pDst, int pan_angles, int pan_rays, float scan_range, float R, float D)
{
	float diagonal = sqrt(1.f * src_width * src_width + src_height * src_height);
	float theta_0 = acos(diagonal / 2.f / R);
	float pan_u0 = D / tan(theta_0);
	float pan_delta_u = (2.f * pan_u0) / pan_rays;
	float angles_separation = scan_range / pan_angles;
	int center_y = (src_height + 1) / 2;
	for (int i = 0; i < pan_angles; ++i)
	{
		int Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight;
		float rad = i * angles_separation;
		float *pCur = ComputeRotatedValues(pSrc, src_width, src_height, rad, Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight);
		for (int j = 0; j < pan_rays; ++j)
		{
			float u = -pan_u0 + j * pan_delta_u;
			float k = -u / D;
			float c = -R * k + center_y;
			pDst[i + j * pan_angles] = LineIntegrate(pCur, Xmin, Ymin, Xmax, Ymax, NewWidth, NewHeight, k, c);
		}
		SAFE_DELETE(pCur);
	}
}


float* ComputeRotatedValues(float *_ptrT, int Width, int Height, float angle, int &Xmin, int &Ymin, int &Xmax, int &Ymax, int &NewWidth, int &NewHeight) 
{
	// ԭʼͼ���ĸ����������
	float x1, x2, x3, x4, y1, y2, y3, y4;
	x1 = 0;							y1 = 0;
	x2 = float(Width - 1);			y2 = 0;
	x3 = x2;						y3 = float(Height - 1);
	x4 = x1;						y4 = y3;
	// �ĸ�����˳ʱ����ת,������ԭ��
	float m_Ox = Width / 2.f;
	float m_Oy = Height / 2.f;
	PositionTransform(x1, y1, angle, m_Ox, m_Oy);
	PositionTransform(x2, y2, angle, m_Ox, m_Oy);
	PositionTransform(x3, y3, angle, m_Ox, m_Oy);
	PositionTransform(x4, y4, angle, m_Ox, m_Oy);
	Xmax = int(FindMaxBetween4Numbers(x1, x2, x3, x4));
	Ymax = int(FindMaxBetween4Numbers(y1, y2, y3, y4));
	Xmin = int(FindMinBetween4Numbers(y1, y2, y3, y4));
	Ymin = int(FindMinBetween4Numbers(y1, y2, y3, y4));
	// ��ͼ���ȡ��߶ȡ�ÿ���ֽ����ı仯
	NewWidth = Xmax - Xmin + 1;
	NewHeight = Ymax - Ymin + 1;
	float *pDst = new float[NewWidth * NewHeight];
	memset(pDst, 0, NewWidth * NewHeight * sizeof(float));
	
	for (int i = 0; i < NewWidth; ++i)
	{
		for (int j = 0; j < NewHeight; ++j)
		{
			float x = float(i + Xmin);
			float y = float(j + Ymin);
			PositionTransform(x, y, -angle, m_Ox, m_Oy);
			*(pDst + i + j * NewWidth) = biLinearInterp(_ptrT, Height, Width, x, y);
		}
	}
	
	return pDst;
}


float GetPositionValue(float *_ptrT, int row, int col, int x, int y)
{
	if (x < 0 || x >= col || y < 0 || y >= row)
		return 0;
	return (float)*(_ptrT + x + y * col);
}


float biLinearInterp(float *_ptrT, int row, int col, float x, float y)
{
	int x1, x2, x3, x4, y1, y2, y3, y4;
	float Ans1, Ans2;
	x1 = floor(x);	y1 = floor(y);
	x2 = x1 + 1;	y2 = y1;
	x3 = x2;		y3 = y1 + 1;
	x4 = x1;		y4 = y3;
	Ans1 = GetPositionValue(_ptrT, row, col, x1, y1) * (1 - x + x1) + GetPositionValue(_ptrT, row, col, x2, y2) * (x - x1);
	Ans2 = GetPositionValue(_ptrT, row, col, x4, y4) * (1 - x + x4) + GetPositionValue(_ptrT, row, col, x3, y3) * (x - x4);
	return (Ans1 * (1 - y + y1) + Ans2 * (y - y1));
}


float cudaLineGetXValue(float &k, float &c, float y)
{
	return (y - c) / k;
}


float cudaLineGetYValue(float &k, float &c, float x)
{
	return (k * x + c);
}


/*
float* pSrc		ͼ������
int Xmin		���½�x����
int Ymin		���½�y����
int Xmax		���Ͻ�x����
int Ymax		���Ͻ�y����
int Width		ͼ����
int Height		ͼ��߶�
float &k		ֱ��б��
float &c		ֱ����y��ؾ�
*/
float LineIntegrate(float* pSrc, int Xmin, int Ymin, int Xmax, int Ymax, int Width, int Height, float &k, float &c) 
{
	vector <CIntSection> Sections;

	for (int i = Xmin; i <= Xmax; ++i)
	{
		float x = i;
		float y = cudaLineGetYValue(k, c, x);
		// ��Ϊֱ���ǵ������������Ե�y����ͼ���˾�break
		if (y < Ymin || y > Ymax)
			break;
		Sections.push_back(CIntSection(x, y));
	}
	// �������ۣ����ټ�����
	if (k >= 0)
	{
		int Y_start = int(c);
		int Y_end = Ymax;

		for (int j = Y_start; j <= Y_end; ++j)
		{
			float y = j;
			float x = cudaLineGetXValue(k, c, y);
			// ���ﲻ��ֱ��break��������ͼ���Կ���
			if (x < Xmin || x > Xmax)
				continue;
			Sections.push_back(CIntSection(x, y));
		}
	}
	else
	{
		int Y_start = int(c);
		int Y_end = Ymin;

		for (int j = Y_start; j >= Y_end; --j)
		{
			float y = j;
			float x = cudaLineGetXValue(k, c, y);
			// ���ﲻ��ֱ��break��������ͼ���Կ���
			if (x < Xmin || x > Xmax)
				continue;
			Sections.push_back(CIntSection(x, y));
		}
	}

	// ����
	sort(Sections.begin(), Sections.end());

	float s = 0.f;
	int Num = Sections.size() - 1;
	for (int i = 0; i < Num; ++i)
	{
		int x = Sections[i].m_xPos - Xmin;
		int y = Sections[i].m_yPos - Ymin;
		if (0 <= x && x < Width && 0 <= y && y < Height)
			s += Distance(Sections[i], Sections[i + 1]) * pSrc[x + y * Width];
	}
	return s;
}