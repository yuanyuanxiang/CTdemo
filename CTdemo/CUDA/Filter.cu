#include "hip/hip_runtime.h"
#include "Filter.h"
#include "hipblas.h"
#include "hipblas.h"

/*
// ******************* �˲� ��ʵ�� ***************** //
   ********** �׶�ʦ����ѧ ������ʵ���� ********** //
// *************** Ԭ���飬2015��4�� *************** //
*/


/*
	g(r, fai) = Fourier(radon(r, fai)) * HW(r)
	radon�任�ĸ���Ҷ�任���˲����Ƕ�ά����Ҷ�任.
*/


#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }


// �������Ⱦ���ת��Ϊ�����Ⱦ���
/*
float *d_pSrc		�����Ⱦ���
float *d_temp		�м�����
int row				�������
int col				�������
*/
__global__ void cudaRowMajor2ColMajor(float *d_pSrc, float *d_temp, int row, int col)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; //i�У����temp
	int j = blockIdx.y * blockDim.y + threadIdx.y; //j�У����temp
	if (i < row && j < col)
		d_pSrc[i + j * row] = d_temp[j + i * col];
}


// �������Ⱦ���ת��Ϊ�����Ⱦ���
/*
float *d_pSrc		�����Ⱦ���
float *d_temp		�м�����
int row				�������
int col				�������
*/
__global__ void cudaColMajor2RowMajor(float *d_pSrc, float *d_temp, int row, int col)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; //i�У����d_pSrc
	int j = blockIdx.y * blockDim.y + threadIdx.y; //j�У����d_pSrc
	if (i < row && j < col)
		d_pSrc[j + i * col] = d_temp[i + j * row];
}


// ��ʼ�������
/*
float* d_Conv		�����(������)
int row				����˵���
int col				����˵���
float w0			��ֹƵ��
*/
__global__ void cudaInitConvolute(float* d_pConv, int row, int col, float w0)
{
	int m = blockIdx.x * blockDim.x + threadIdx.x; //m��
	int n = blockIdx.y * blockDim.y + threadIdx.y; //n��
	if (m < row && n < col)
		d_pConv[m + n * row] = cudaConvKernel(m - n, w0);
}


// ����ˣ�HW(r). w0 - ��Ƶ��ֹ
__device__ float cudaConvKernel(float x, float w0)
{
	if(x == 0) return w0 * w0;
	float theta = 2 * PI * w0 * x;
	return (-1 + cos(theta) + theta * sin(theta)) / (2 * PI * PI * x * x);
}


// RL�˲���: w0 = 1 / (2d)
__device__ float cudaRLFilter(int n, float d)
{
	if (n == 0) return 1.f / (4 * d * d);
	if (n % 2 == 0) return 0.f;
	return -1.f / (PI * PI * n * n * d * d);
}


// �������ͶӰ���������
/*
float* h_pDst				Ŀ��
float* prj					ָ��ͶӰ���ݵ�ָ��
int row						ͶӰ���ݵ��У��˴���ʾr
int col						ͶӰ���ݵ��У��˴���ʾfai
float delta_r				��ɢ��֮���dr
float w0					��Ƶ��ֹ
*/
__host__ const char* cudaConvolute(float* h_pDst, float* h_prj, int row, int col, float delta_r, float w0)
{
	hipError_t error;
	const char* errstr = 0;
	hipDeviceProp_t deviceProp;
	int devID = 0;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, errstr);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, errstr);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, errstr);
	// ����������� < 1.1
	if (deviceProp.major < 1) return errstr;
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return errstr;

	float *d_prj = NULL, *d_pDst = NULL, *d_temp = NULL;
	size_t length = row * col * sizeof(float);

	error = hipMalloc((void **)&d_prj, length);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_pDst, length);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_temp, length);
	CHECK_ERRORS(error, errstr);
	error = hipMemset(d_pDst, 0, length);
	CHECK_ERRORS(error, errstr);

	cublasInit();
	float *d_pConv = NULL;
	error = hipMalloc((void **)&d_pConv, row * row * sizeof(float));
	CHECK_ERRORS(error, errstr);
	dim3 dimBlock(16, 16);
	dim3 dimGrid((row + dimBlock.x - 1) / dimBlock.x, (row + dimBlock.y - 1) / dimBlock.y);
	cudaInitConvolute<<<dimGrid, dimBlock>>>(d_pConv, row, row, w0);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);
	// �������Ⱦ���ת��Ϊ�����Ⱦ���
	error = hipMemcpy(d_temp, h_prj, length, hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	cudaRowMajor2ColMajor<<<dimGrid, dimBlock>>>(d_prj, d_temp, row, col);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);

	float alpha = 1.f, belta = 0.f;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	//P = C * P -> P' = P' * C';
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, row, col, row, &alpha, d_pConv, row, d_prj, row, &belta, d_pDst, row);
	// �������Ⱦ���ת��Ϊ�����Ⱦ���
	cudaColMajor2RowMajor<<<dimGrid, dimBlock>>>(d_temp, d_pDst, row, col);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(h_pDst, d_temp, length, hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, errstr);

	hipFree(d_pDst);
	hipFree(d_prj);
	hipFree(d_pConv);
	hipFree(d_temp);
	hipblasDestroy(handle);
	cublasShutdown();
	hipDeviceReset();
	return errstr;
	/* ������ʽ��P(m, n) = C(m, m) * P(m, n)
	for (int m = 0; m < row; ++m)
	{
		for (int n = 0; n < col; ++n)
		{
			for (int i = 0; i < row; ++i)
				h_pDst[n + m * col] += h_prj[n + i * col] * ConvKernel(m - i, w0);
		}
	}*/
}