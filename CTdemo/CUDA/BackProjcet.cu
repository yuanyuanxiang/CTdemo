#include "hip/hip_runtime.h"
#include "BackProject.h"


/*
// ******************* �˲� ��ʵ�� ***************** //
   ********** �׶�ʦ����ѧ ������ʵ���� ********** //
// *************** Ԭ���飬2015��4�� *************** //
*/


#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }


// �����Ǻ���ֵ���г�ʼ��
__global__ void cudaInitCosSin(float *cos_theta, float *sin_theta, float delta_fai, int angles)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < angles)
	{
		float theta = i * delta_fai;
		cos_theta[i] = cos(theta);
		sin_theta[i] = sin(theta);
	}
}


// ��ͶӰ�ĺ˺���(���˺�����forѭ�������Ƚϸߵ�ʱ���Կ���������ԭ����)
/*
ǰ8��������BackProject�Ĳ���һ����
int cx		ͼ���������꣬������½�
int cy		ͼ���������꣬������½�
int med		���ߵ��������꣬�̶��ĽǶ�
*/
__global__ void cudaBackProjectKernel(float* d_pDst, float* d_prj, int width, int height, int rays, int angles, float *cos_fai, float *sin_fai, float delta_r, float delta_fai, 
								  int cx, int cy, int med)
{
	int m = blockIdx.x * blockDim.x + threadIdx.x; //i��
	int n = blockIdx.y * blockDim.y + threadIdx.y; //j��

	if(m < height && n < width)
	{
		for (int i = 0; i < angles; ++i)
		{
			float r = (n - cx) * cos_fai[i] + (m - cy) * sin_fai[i];
			d_pDst[n + m * width] += cudaLinearInterp(med + r * delta_r, i, d_prj, angles, rays);
		}
		d_pDst[n + m * width] *= delta_fai;
	}
}


// ��ͶӰ
/*
float* h_pDst		Ŀ��
float* prj			������ͶӰ
int width			�ؽ�ͼ��Ŀ�
int height			�ؽ�ͼ��ĸ�
int rays			ͶӰ������(r)
int angles			ͶӰ������(fai)
float delta_r		�����ܶ�
float delta_fai		ɨ�����
*/
__host__ const char* cudaBackProject(float* h_pDst, float* h_prj, int width, int height, int rays, int angles, float delta_r, float delta_fai)
{
	hipError_t error;
	const char* errstr = 0;
	hipDeviceProp_t deviceProp;
	int devID = 0;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, errstr);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, errstr);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, errstr);
	// ����������� < 1.1
	if (deviceProp.major < 1) return "��������С��1.0.";
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return "��������С��1.1.";

	float *d_pDst = NULL;
	float *d_prj = NULL;
	error = hipMalloc((void **)&d_pDst, width * height * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pDst, h_pDst, width * height * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_prj, rays * angles * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_prj, h_prj, rays * angles * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);

	int cx = (width + 1) / 2;
	int cy = (height + 1) / 2;
	int med = (rays + 1) / 2;

	// 2015.5.19 �����Ǻ����ŵ�ѭ��������
	float *cos_fai = NULL;
	float *sin_fai = NULL;
	error = hipMalloc((void **)&cos_fai, angles * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&sin_fai, angles * sizeof(float));
	CHECK_ERRORS(error, errstr);

	int ThreadsNum = 256;
	cudaInitCosSin<<<(angles + ThreadsNum - 1) / ThreadsNum, ThreadsNum>>>(cos_fai, sin_fai, delta_fai, angles);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);

	dim3 dimBlock(16, 16);
	dim3 dimGrid((height + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);
	cudaBackProjectKernel<<<dimGrid, dimBlock>>>(d_pDst, d_prj, width, height, rays, angles, cos_fai, sin_fai, delta_r, delta_fai, cx, cy, med);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);

	error = hipMemcpy(h_pDst, d_pDst, width * height * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, errstr);

	hipFree(cos_fai);
	hipFree(sin_fai);
	hipFree(d_pDst);
	hipFree(d_prj);
	hipDeviceReset();
	return errstr;
}


// ��(r, fai)�����в�ֵ��ĳ���ֵ�����Բ�ֵ��
__device__ float cudaLinearInterp(float r_id, int fai_id, float* prj, int angles, int rays)
{
	int r1 = floor(r_id), r2 = r1 + 1;
	if ( r1 < 0 || r2 >= rays)
		return 0;
	return prj[fai_id + r1 * angles] * (1 - r_id + r1) + prj[fai_id + r2 * angles] * (r_id - r1);
}