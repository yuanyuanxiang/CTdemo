#include "hip/hip_runtime.h"
#include "BackProject.h"


/*
// ******************* �˲� ��ʵ�� ***************** //
   ********** �׶�ʦ����ѧ ������ʵ���� ********** //
// *************** Ԭ���飬2015��4�� *************** //
*/


#define CHECK_ERRORS(err, str) if(err != hipSuccess) { str = hipGetErrorString(err); hipDeviceReset(); return str; }

// ��ͶӰ�ĺ˺���(���˺�����forѭ�������Ƚϸߵ�ʱ���Կ���������ԭ����)
/*
ǰ8��������BackProject�Ĳ���һ����
int cx		ͼ���������꣬������½�
int cy		ͼ���������꣬������½�
int med		���ߵ��������꣬�̶��ĽǶ�
*/
__global__ void cudaBackProjectKernel(float* d_pDst, float* d_prj, int width, int height, int rays, int angles, float delta_r, float delta_fai, 
								  int cx, int cy, int med)
{
	int m = blockIdx.x * blockDim.x + threadIdx.x; //i��
	int n = blockIdx.y * blockDim.y + threadIdx.y; //j��
	if(m < height && n < width)
	{
		for (int i = 0; i < angles; ++i)
		{
			float fai = i * delta_fai;
			float r = (n - cx) * cos(fai) + (m - cy) * sin(fai);
			d_pDst[n + m * width] += cudaLinearInterp(med + r / delta_r, i, d_prj, angles, rays);
		}
		d_pDst[n + m * width] *= delta_fai;
	}
}


// ��ͶӰ
/*
float* h_pDst		Ŀ��
float* prj			������ͶӰ
int width			�ؽ�ͼ��Ŀ�
int height			�ؽ�ͼ��ĸ�
int rays			ͶӰ������(r)
int angles			ͶӰ������(fai)
float delta_r		ɨ�����
float delta_fai		ɨ�����
*/
__host__ const char* cudaBackProject(float* h_pDst, float* h_prj, int width, int height, int rays, int angles, float delta_r, float delta_fai)
{
	hipError_t error;
	const char* errstr = 0;
	hipDeviceProp_t deviceProp;
	int devID = 0;
	error = hipGetDeviceCount(&devID);
	CHECK_ERRORS(error, errstr);
	// pick the device with highest Gflops/s
	devID = gpuGetMaxGflopsDeviceId();
	error = hipSetDevice(devID);
	CHECK_ERRORS(error, errstr);
	error = hipGetDeviceProperties(&deviceProp, devID);
	CHECK_ERRORS(error, errstr);
	// ����������� < 1.1
	if (deviceProp.major < 1) return errstr;
	else if (deviceProp.major == 1 && deviceProp.minor < 1) return errstr;

	float *d_pDst = NULL;
	float *d_prj = NULL;
	error = hipMalloc((void **)&d_pDst, width * height * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_pDst, h_pDst, width * height * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);
	error = hipMalloc((void **)&d_prj, rays * angles * sizeof(float));
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(d_prj, h_prj, rays * angles * sizeof(float), hipMemcpyHostToDevice);
	CHECK_ERRORS(error, errstr);

	int cx = (width + 1) / 2;
	int cy = (height + 1) / 2;
	int med = (rays + 1) / 2;

	dim3 dimBlock(16, 16);
	dim3 dimGrid((height + dimBlock.x - 1) / dimBlock.x, (width + dimBlock.y - 1) / dimBlock.y);
	cudaBackProjectKernel<<<dimGrid, dimBlock>>>(d_pDst, d_prj, width, height, rays, angles, delta_r, delta_fai, cx, cy, med);
	error = hipGetLastError();
	CHECK_ERRORS(error, errstr);
	error = hipMemcpy(h_pDst, d_pDst, width * height * sizeof(float), hipMemcpyDeviceToHost);
	CHECK_ERRORS(error, errstr);

	hipFree(d_pDst);
	hipFree(d_prj);
	hipDeviceReset();
	return errstr;
	/*
	for (int m = 0; m < height; ++m)
	{
		for (int n = 0; n < width; ++n)
		{
			x1 = n - cx;
			y1 = m - cy;
			for (int i = 0; i < angles; ++i)
			{
				fai = i * delta_fai;
				r = x1 * cos(fai) + y1 * sin(fai);
				r_id = r / delta_r;
				h_pDst[n + m * width] += LinearInterp(med + r_id, i, h_prj, angles, rays);
			}
			h_pDst[n + m * width] *= delta_fai;
		}
	}
	*/
}


// ��(r, fai)�����в�ֵ��ĳ���ֵ�����Բ�ֵ��
__device__ float cudaLinearInterp(float r_id, int fai_id, float* prj, int angles, int rays)
{
	int r1 = floor(r_id), r2 = r1 + 1;
	if ( r1 < 0 || r2 >= rays)
		return 0.f;
	return prj[fai_id + r1 * angles] * (1 - r_id + r1) + prj[fai_id + r2 * angles] * (r_id - r1);
}